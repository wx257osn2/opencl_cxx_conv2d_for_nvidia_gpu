
#include <hip/hip_runtime.h>
extern "C" __global__ void convolution_general(
    const unsigned char* __restrict__ im,
    int width,
    int height,
    const float* __restrict__ kernel,
    int kernel_size,
    unsigned char* __restrict__ output){
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  const int half_k = kernel_size / 2;
  if(y < half_k
  || height - half_k <= y
  || x < half_k
  || width - half_k <= x)
    return;
  float t = 0.f;
  for(int i = 0; i < kernel_size; ++i)
    for(int j = 0; j < kernel_size; ++j)
      t += im[(y+j-half_k)*width+x+i-half_k] * kernel[i*kernel_size+j];
  output[y*width+x] = static_cast<unsigned char>(min(max(t, 0.f), 255.f));
}
